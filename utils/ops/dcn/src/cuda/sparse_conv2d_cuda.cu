#include "hip/hip_runtime.h"
#include <vector>
#include "cuda/sparse_2d_im2col_cuda.cuh"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// #include <THC/THC.h>
// #include <THC/THCAtomics.cuh>
// #include <THC/THCDeviceUtils.cuh>

// extern THCState *state;

// author: Charles Shang
// https://github.com/torch/cunn/blob/master/lib/THCUNN/generic/SpatialConvolutionMM.cu


at::Tensor
sparse_conv2d_cuda_forward(const at::Tensor &input,
                           const at::Tensor &weight,
                           const at::Tensor &bias,
                           const at::Tensor &offset,
                           const int kernel_h,
                           const int kernel_w,
                           const int stride_h,
                           const int stride_w,
                           const int pad_h,
                           const int pad_w,
                           const int dilation_h,
                           const int dilation_w,
                           const int group,
                           const int deformable_group,
                           const int num_pts,
                           const int im2col_step)
{
    // THCAssertSameGPU(THCudaTensor_checkGPU(state, 5, input, weight, bias, offset, mask));

    AT_ASSERTM(input.is_contiguous(), "input tensor has to be contiguous");
    AT_ASSERTM(weight.is_contiguous(), "weight tensor has to be contiguous");

    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(weight.type().is_cuda(), "weight must be a CUDA tensor");
    AT_ASSERTM(bias.type().is_cuda(), "bias must be a CUDA tensor");
    AT_ASSERTM(offset.type().is_cuda(), "offset must be a CUDA tensor");

    const int batch = input.size(0);
    const int channels = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int channels_kernel = weight.size(1);
    const int kernel_n = weight.size(2);

    const int im2col_step_ = std::min(batch, im2col_step);

    AT_ASSERTM(batch % im2col_step_ == 0, "batch(%d) must divide im2col_step(%d)", batch, im2col_step_)

    AT_ASSERTM((channels % group == 0) && (channels_out % group == 0), 
        "channels(%d) and channels_out(%d) must divide group(%d)", channels, channels_out, group)

    AT_ASSERTM(kernel_n == num_pts, "kernel_n ", kernel_n, " and num_pts ", num_pts, " must match")

    AT_ASSERTM(offset.size(1) == deformable_group * 2 * num_pts, "offset channel ", offset.size(1),
            " must match deformable group ", deformable_group, " and num_pts ", num_pts)

    AT_ASSERTM(channels == (channels_kernel * group),
               "Input shape and kernel channels wont match: (%d vs %d).", channels, channels_kernel * group);

    const int height_out = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

    auto output = at::empty({batch * height_out * width_out, channels_out}, input.options());

    // prepare group weight and bias
    auto weight_g = weight.view({group, channels_out/group, channels_kernel, kernel_n});
    auto bias_g = bias.view({group, channels_out/group});

    // define alias for easy use
    const int batch_n = im2col_step_;
    const int per_input_size = channels * height * width;
    const int per_offset_size = offset.size(1) * offset.size(2) * offset.size(3);
    auto output_n = output.view({batch/im2col_step_, batch_n * height_out * width_out, channels_out});
    for (int n = 0; n < batch/im2col_step_; ++n)
    {
        auto columns = at::empty({channels * kernel_n, batch_n * height_out * width_out}, input.options());
        AT_DISPATCH_FLOATING_TYPES(input.type(), "sparse_conv2d_forward_cuda", ([&] {
            sparse_2d_im2col_cuda(at::cuda::getCurrentCUDAStream(),
                                  input.data<scalar_t>() + n * im2col_step_ * per_input_size,
                                  offset.data<scalar_t>() + n * im2col_step_ * per_offset_size,
                                  batch_n, channels, height, width,
                                  height_out, width_out, kernel_h, kernel_w,
                                  pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
                                  deformable_group, num_pts, columns.data<scalar_t>());

        }));

        auto columns_g = columns.view({group, channels/group * kernel_n, batch_n * height_out * width_out});
        auto output_g = output_n.select(0, n).view({batch_n * height_out * width_out, group, channels_out/group});
        for (int g = 0; g < group; ++g)
        {
            auto columns_gm = columns_g.select(0, g).t();
            auto weight_gm = weight_g.select(0, g).view({channels_out/group, channels_kernel * kernel_n}).t();
            auto output_m = at::addmm(bias_g.select(0, g), columns_gm, weight_gm);
            output_g.select(1, g) = output_m.view({batch_n * height_out * width_out, channels_out/group});
        }

    }

    output = output.view({batch, height_out, width_out, channels_out}).permute({0, 3, 1, 2}).contiguous();

    return output;
}

std::vector<at::Tensor> sparse_conv2d_cuda_backward(const at::Tensor &input,
                                                    const at::Tensor &weight,
                                                    const at::Tensor &bias,
                                                    const at::Tensor &offset,
                                                    const at::Tensor &grad_output,
                                                    const int kernel_h,
                                                    const int kernel_w,
                                                    const int stride_h,
                                                    const int stride_w,
                                                    const int pad_h,
                                                    const int pad_w,
                                                    const int dilation_h,
                                                    const int dilation_w,
                                                    const int group,
                                                    const int deformable_group,
                                                    const int num_pts,
                                                    const int im2col_step)
{

    AT_ASSERTM(input.is_contiguous(), "input tensor has to be contiguous");
    AT_ASSERTM(weight.is_contiguous(), "weight tensor has to be contiguous");

    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(weight.type().is_cuda(), "weight must be a CUDA tensor");
    AT_ASSERTM(bias.type().is_cuda(), "bias must be a CUDA tensor");
    AT_ASSERTM(offset.type().is_cuda(), "offset must be a CUDA tensor");

    const int batch = input.size(0);
    const int channels = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int channels_kernel = weight.size(1);
    const int kernel_n = weight.size(2);

    const int batch_ = grad_output.size(0);
    const int channels_out_ = grad_output.size(1);
    const int height_out_ = grad_output.size(2);
    const int width_out_ = grad_output.size(3);

    const int im2col_step_ = std::min(im2col_step, batch);

    AT_ASSERTM(batch % im2col_step_ == 0, "batch(%d) must divide im2col_step(%d)", batch, im2col_step_)

    AT_ASSERTM((channels % group == 0) && (channels_out % group == 0), 
        "channels(%d) and channels_out(%d) must divide group(%d)", channels, channels_out, group)

    AT_ASSERTM(kernel_n == num_pts, "kernel_n ", kernel_n, " and num_pts ", num_pts, " must match")

    AT_ASSERTM(offset.size(1) == deformable_group * 2 * num_pts, "offset channel ", offset.size(1),
               " must match deformable group ", deformable_group, " and num_pts ", num_pts)

    AT_ASSERTM(channels == (channels_kernel * group),
               "Input shape and kernel channels wont match: (%d vs %d).", channels, channels_kernel * group);

    const int height_out = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

    AT_ASSERTM(batch == batch_,
               "Input shape and grad_out batch wont match: (%d vs %d).", batch, batch_);

    AT_ASSERTM(channels_out == channels_out_,
               "Input shape and grad_out channels_out wont match: (%d vs %d).", channels_out, channels_out_);

    AT_ASSERTM(height_out == height_out_ && width_out == width_out_,
               "Input shape and grad_out shape wont match: (%d x %d vs %d x %d).", height_out, height_out_, width_out, width_out_);

    auto grad_input = at::zeros_like(input);
    auto grad_offset = at::zeros_like(offset);
    auto grad_weight = at::zeros_like(weight);
    auto grad_bias = at::zeros_like(bias);

    // prepare group weight and bias
    auto weight_g = weight.view({group, channels_out/group, channels_kernel, kernel_n});
    auto grad_weight_g = grad_weight.view({group, channels_out/group, channels_kernel, kernel_n});
    auto grad_bias_g = grad_bias.view({group, channels_out/group});

    const int batch_n = im2col_step_;
    const int per_input_size = channels * height * width;
    const int per_offset_size = offset.size(1) * offset.size(2) * offset.size(3);
    auto grad_output_n = grad_output.view({batch/im2col_step_, batch_n, channels_out, height_out, width_out});
    for (int n = 0; n < batch/im2col_step_; ++n)
    {
        auto grad_output_g = grad_output_n.select(0, n).view({batch_n, group, channels_out/group, height_out, width_out});
        auto ones = at::ones({batch_n * height_out * width_out}, input.options());
        auto columns = at::empty({channels * kernel_n, batch_n * 1 * height_out * width_out}, input.options());
        auto columns_g = columns.view({group, channels/group * kernel_n, batch_n * height_out * width_out});
        for (int g = 0; g < group; ++g)
        {
            auto grad_output_gm = grad_output_g.select(1, g).permute({1, 0, 2, 3}).contiguous().view({channels_out/group, batch_n * height_out * width_out});
            auto weight_gm = weight_g.select(0, g).view({channels_out/group, channels_kernel * kernel_n}).t();
            columns_g.select(0, g) = at::mm(weight_gm, grad_output_gm);
        }

        AT_DISPATCH_FLOATING_TYPES(input.type(), "sparse_conv2d_backward_cuda", ([&] {
            sparse_2d_col2im_coord_cuda(at::cuda::getCurrentCUDAStream(),
                                        columns.data<scalar_t>(),
                                        input.data<scalar_t>() + n * im2col_step_ * per_input_size,
                                        offset.data<scalar_t>() + n * im2col_step_ * per_offset_size,
                                        batch_n, channels, height, width,
                                        height_out, width_out, kernel_h, kernel_w,
                                        pad_h, pad_w, stride_h, stride_w,
                                        dilation_h, dilation_w,
                                        deformable_group, num_pts,
                                        grad_offset.data<scalar_t>() + n * im2col_step_ * per_offset_size);
            // gradient w.r.t. input data
            sparse_2d_col2im_cuda(at::cuda::getCurrentCUDAStream(),
                                  columns.data<scalar_t>(),
                                  offset.data<scalar_t>() + n * im2col_step_ * per_offset_size,
                                  batch_n, channels, height, width,
                                  height_out, width_out, kernel_h, kernel_w,
                                  pad_h, pad_w, stride_h, stride_w,
                                  dilation_h, dilation_w,
                                  deformable_group, num_pts,
                                  grad_input.data<scalar_t>() + n * im2col_step_ * per_input_size);

            // gradient w.r.t. weight, dWeight should accumulate across the batch and group
            sparse_2d_im2col_cuda(at::cuda::getCurrentCUDAStream(),
                                  input.data<scalar_t>() + n * im2col_step_ * per_input_size,
                                  offset.data<scalar_t>() + n * im2col_step_ * per_offset_size,
                                  batch_n, channels, height, width,
                                  height_out, width_out, kernel_h, kernel_w,
                                  pad_h, pad_w, stride_h, stride_w,
                                  dilation_h, dilation_w,
                                  deformable_group, num_pts,
                                  columns.data<scalar_t>());

        }));

        for (int g = 0; g < group; ++g)
        {
            auto grad_output_gm = grad_output_g.select(1, g).permute({1, 0, 2, 3}).contiguous().view({channels_out/group, batch_n * height_out * width_out});
            auto columns_gm = columns_g.select(0, g).t();
            auto grad_weight_gm = grad_weight_g.select(0, g).view({channels_out/group, channels_kernel * kernel_n});
            auto grad_bias_gm = grad_bias_g.select(0, g);
            grad_weight_g.select(0, g) = at::addmm(grad_weight_gm, grad_output_gm, columns_gm).view_as(grad_weight_g.select(0, g));
            grad_bias_g.select(0, g) = at::addmv(grad_bias_gm, grad_output_gm, ones);
        }

    }

    return {
        grad_input, grad_offset, grad_weight, grad_bias
    };
}